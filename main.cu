#include <iostream>
#include <hip/hip_runtime.h>

using std::cout;
using std::endl;

void printGPUInfo() {
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess) {
        std::cerr << "hipGetDeviceCount returned " << static_cast<int>(error_id) << " -> " << hipGetErrorString(error_id) << std::endl;
        return;
    }

    if (deviceCount == 0) {
        std::cout << "There are no available CUDA devices." << std::endl;
    } else {
        std::cout << "Detected " << deviceCount << " CUDA Capable device(s)." << std::endl;
    }

    for (int dev = 0; dev < deviceCount; ++dev) {
        hipSetDevice(dev);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout << "Device " << dev << ": " << deviceProp.name << std::endl;
        std::cout << "  CUDA Capability Major/Minor version number: " << deviceProp.major << "." << deviceProp.minor << std::endl;
        std::cout << "  Total amount of global memory: " << deviceProp.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
        std::cout << "  Number of multiprocessors: " << deviceProp.multiProcessorCount << std::endl;
        std::cout << "  Total amount of constant memory: " << deviceProp.totalConstMem << " bytes" << std::endl;
        std::cout << "  Total amount of shared memory per block: " << deviceProp.sharedMemPerBlock << " bytes" << std::endl;
        std::cout << "  Total number of registers available per block: " << deviceProp.regsPerBlock << std::endl;
    }
}
int
main()
{
    cout << "Transformer Model C++ Basic Implementation" << endl;
    printGPUInfo();

    return 0;
}
