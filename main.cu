#include <iostream>
#include <vector>
#include "InputEmbeddings.h"
#include "PositionalEncoding.h"

using std::cout;
using std::endl;

void printGPUInfo() {
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess) {
        std::cerr << "hipGetDeviceCount returned " << static_cast<int>(error_id) << " -> " << hipGetErrorString(error_id) << std::endl;
        return;
    }

    if (deviceCount == 0) {
        std::cout << "There are no available CUDA devices." << std::endl;
    } else {
        std::cout << "Detected " << deviceCount << " CUDA Capable device(s)." << std::endl;
    }

    for (int dev = 0; dev < deviceCount; ++dev) {
        hipSetDevice(dev);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout << "Device " << dev << ": " << deviceProp.name << std::endl;
        std::cout << "  CUDA Capability Major/Minor version number: " << deviceProp.major << "." << deviceProp.minor << std::endl;
        std::cout << "  Total amount of global memory: " << deviceProp.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
        std::cout << "  Number of multiprocessors: " << deviceProp.multiProcessorCount << std::endl;
        std::cout << "  Total amount of constant memory: " << deviceProp.totalConstMem << " bytes" << std::endl;
        std::cout << "  Total amount of shared memory per block: " << deviceProp.sharedMemPerBlock << " bytes" << std::endl;
        std::cout << "  Total number of registers available per block: " << deviceProp.regsPerBlock << std::endl;
    }
}

void printTensor(const float* tensor, int batch_size, int seq_len, int d_model) {
    for (int b = 0; b < batch_size; ++b) {
        std::cout << "Batch " << b << ":" << std::endl;
        for (int s = 0; s < seq_len; ++s) {
            std::cout << "  Seq " << s << ": ";
            for (int d = 0; d < d_model; ++d) {
                std::cout << tensor[b * seq_len * d_model + s * d_model + d] << " ";
            }
            std::cout << std::endl;
        }
    }
}

int 
main() 
{
    cout << "Transformer Model C++ Basic Implementation" << endl;
    printGPUInfo();

    // Parameters
    // Multihead 
    int d_model = 4;
    // Size of dict
    int vocab_size = 10;
    // input will have...
    int batch_size = 2;
    // length of input
    int seq_len = 3;
    // likelyhood of dropout
    float dropout = 0.1;

    InputEmbeddings embeddings(d_model, vocab_size);
    PositionalEncoding positional_encoding(d_model, seq_len, dropout);

    std::vector<int> input = {1, 2, 3, 4, 5, 6};

    float* d_output;   
    checkCudaErrors(hipMalloc(&d_output, batch_size * seq_len * d_model * sizeof(float)));

    embeddings.forward(input.data(), d_output, batch_size, seq_len);
    positional_encoding.forward(d_output, batch_size, seq_len);

    std::vector<float> output(batch_size * seq_len * d_model);
    checkCudaErrors(hipMemcpy(output.data(), d_output, output.size() * sizeof(float), hipMemcpyDeviceToHost));

    printTensor(output.data(), batch_size, seq_len, d_model);

    return 0;
}
