#include "hip/hip_runtime.h"
#include "Utilities.h"
#include <iostream>

void 
checkCudaErrors(hipError_t err) 
{
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        exit(err);
    }
}

void 
checkCublasErrors(hipblasStatus_t status) 
{
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "cuBLAS Error: " << status << std::endl;
        exit(status);
    }
}

__global__ void 
dropoutKernel(float* x, float* random_values, int size, float dropout) 
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        if (random_values[idx] < dropout) {
            x[idx] = 0.0f;
        }
    }
}


