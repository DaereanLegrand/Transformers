#include "Utilities.h"
#include <iostream>

void 
checkCudaErrors(hipError_t err) 
{
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        exit(err);
    }
}

void 
checkCublasErrors(hipblasStatus_t status) 
{
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "cuBLAS Error: " << status << std::endl;
        exit(status);
    }
}
